#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void testKernel(char *s) {
	int i = threadIdx.x;
	
	memcpy(s + i + 5, s + i, 1);
	assert(1);
}

bool test_cudaHostAlloc() {
	bool res = true;
	char *dptr1, *hptr1;

	// First way: using hipHostAlloc()
	// Alloc host page-locked memory
	checkCudaErrors(hipHostAlloc(&hptr1, 10, hipHostMallocMapped));

	// Get corresponding device pointer
	checkCudaErrors(hipHostGetDevicePointer(&dptr1, hptr1, 0));

	// Initialize this memory
	for(int i = 0; i < 5; i++)
		hptr1[i] = i;
	for(int i = 5; i < 10; i++)
		hptr1[i] = 0;

	// Test kernel
	testKernel<<<1, 5>>>(dptr1);
	hipDeviceSynchronize();

	// Check result
	for(int i = 0; i < 10; i++){
		printf("s[%d]: %d\n", i, hptr1[i]);
		res = (hptr1[i] == i % 5) ? res : false;
	}

	// Free memory
	checkCudaErrors(hipHostFree(hptr1));

	return res;
}

bool test_cudaHostRegister() {
	bool res = true;
	char *dptr1, *hptr1;

	// Second way: using hipHostRegister()
	// Alloc host memory
	hptr1 = new char[10];
	assert(hptr1);

	// Page-lock host memory
	hipHostRegister(hptr1, 10, hipHostRegisterMapped);

	// Get corresponding device pointer
	checkCudaErrors(hipHostGetDevicePointer(&dptr1, hptr1, 0));

	// Initialize this memory
	for(int i = 0; i < 5; i++)
		hptr1[i] = i;
	for(int i = 5; i < 10; i++)
		hptr1[i] = 0;

	// Test kernel
	if(hipDeviceAttributeCanUseHostPointerForRegisteredMem != 0){
		printf("Can directly use host pointer to substitute device pointer on this machine.\n");
		testKernel<<<1, 5>>>(hptr1);
	}
	else{
		printf("This machine does not support substituting host pointer for device pointer.\n");
		testKernel<<<1, 5>>>(dptr1);
	}
	hipDeviceSynchronize();

	// Check result
	for(int i = 0; i < 10; i++){
		printf("s[%d]: %d\n", i, hptr1[i]);
		res = (hptr1[i] == i % 5) ? res : false;
	}

	// Free memory
	hipHostUnregister(hptr1);
	delete hptr1;

	return res;
}

void start_test() {
	// Enable memory mapping
	hipSetDeviceFlags(hipDeviceMapHost);

	if(test_cudaHostAlloc())
		printf("hipHostAlloc(): PASS\n");
	else
		printf("hipHostAlloc(): ERROR\n");

	if(test_cudaHostRegister())
		printf("hipHostRegister(): PASS\n");
	else
		printf("hipHostRegister(): ERROR\n");
}

int main() {
	start_test();

	return 0;
}