#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "nf.cuh"

using namespace std;

#define THREADPERBLOCK	256
#define SHARE_MEM_SIZE  512
#define MAX_PKT_SIZE 64

struct PKT{
	char pkt[MAX_PKT_SIZE];
};

__global__ void gpu_nf_logic(char* pkt_batch, char *state_batch, char *extra_info, int flowDim, int nflows) {

	__shared__ struct ips_flow_state gpu_ips_flow_state[32];

	
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if(id >= nflows) return ;

	PKT*pkts =(PKT*)pkt_batch + id * flowDim;
	struct ips_flow_state* state_ptr=(struct ips_flow_state*)state_batch;


	// for(int i= 0 ;i <DFA_NUM; i++){
	// 	gpu_ips_flow_state[id%32]._state[i]= state_ptr[id]._state[i];
	// 	gpu_ips_flow_state[id%32]._dfa_id[i] = state_ptr[id]._dfa_id[i];
	// 	gpu_ips_flow_state[id%32]._alert[i] = state_ptr[id]._alert[i];

	// }
	
	
	for(int i = 0; i < flowDim; i++) {
		if(pkts[i].pkt[0] == 0) {
			int j;
			for(j = 1; j < 14; j++) {
				if(pkts[i].pkt[j] == 0)
					break;
			}
			if(j != 14) // the whole Ethernet header is empty, means a empty packet, break the loop
				break;
		}
 			
		//NF::nf_logic((char*)pkts[i].pkt, &gpu_ips_flow_state[id%32], ((struct gpu_IPS *)extra_info)->dfa_arr);
		NF::nf_logic((char*)pkts[i].pkt, &state_ptr[id], ((struct gpu_IPS *)extra_info)->dfa_arr);
	}
	
	// for(int i= 0 ;i <DFA_NUM; i++){
	// 	state_ptr[id]._state[i]= gpu_ips_flow_state[id%32]._state[i];
	// 	state_ptr[id]._dfa_id[i]= gpu_ips_flow_state[id%32]._dfa_id[i];
	// 	state_ptr[id]._alert[i] = gpu_ips_flow_state[id%32]._alert[i];

	// }

	return;	
	

}

void gpu_launch(char *pkt_batch, char *state_batch, char *extra_info, int flowDim, int nflows,hipStream_t stream) {
	// Calculate block amounts
	assert(nflows > 0);
	int nblocks = (nflows + THREADPERBLOCK - 1) / THREADPERBLOCK;
//printf("nblocks = %d, nthread = %d, nflows = %d\n", nblocks, THREADPERBLOCK, nflows);
	gpu_nf_logic<<<nblocks, THREADPERBLOCK, SHARE_MEM_SIZE, stream>>>(pkt_batch, state_batch, extra_info, flowDim, nflows);
	//gpu_nf_logic<<<1, 1, SHARE_MEM_SIZE, stream>>>(pkt_batch, state_batch, extra_info, flowDim, nflows);
}

void gpu_sync(hipStream_t stream) {
	checkCudaErrors(hipStreamSynchronize(stream));
}

void gpu_mem_map(void *ptr, size_t size) {
	checkCudaErrors(hipHostRegister(ptr, size, hipHostRegisterMapped));
}

void gpu_mem_unmap(void *ptr) {
	checkCudaErrors(hipHostUnregister(ptr));
}

void gpu_malloc(void** devPtr, size_t size){
	checkCudaErrors(hipMalloc(devPtr, size));
}

void gpu_malloc_host(void** devPtr, size_t size){
	checkCudaErrors(hipHostAlloc(devPtr, size,hipHostMallocMapped));
}


void gpu_memcpy_async_h2d(void* dst, const void*src, size_t count ,hipStream_t stream=0){
	checkCudaErrors(hipMemcpyAsync(dst,src,count,hipMemcpyHostToDevice,stream));
}

void gpu_memcpy_async_d2h(void* dst, const void*src, size_t count, hipStream_t stream=0 ){
	checkCudaErrors(hipMemcpyAsync(dst,src,count,hipMemcpyDeviceToHost,stream));
	
}


void gpu_memset_async(void * devPtr, int value, size_t count, hipStream_t stream = 0){

	checkCudaErrors(hipMemsetAsync(devPtr,value,count,stream))	;
}

void gpu_free(void* devPtr){
	checkCudaErrors(hipFree(devPtr));
}


void gpu_free_host(void* devPtr){
	checkCudaErrors(hipHostFree(devPtr));
}




void create_stream(hipStream_t* stream_ptr){

	checkCudaErrors(hipStreamCreateWithFlags(stream_ptr,hipStreamNonBlocking));
}

void destory_stream(hipStream_t stream){

checkCudaErrors(hipStreamDestroy(stream));
}

void create_event(hipEvent_t* event_ptr){

checkCudaErrors(hipEventCreateWithFlags(event_ptr,hipEventDisableTiming));
}


